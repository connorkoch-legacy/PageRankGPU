
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <cstring>
#include <sys/time.h>
#include <vector>
#include <map>
using namespace std;

__global__ void pagerank(int* rowPtrs, int* colIdx, double* values, double* ranks, double* newRanks, int numRows) {

    unsigned int row = blockIdx.x*blockDim.x + threadIdx.x;

    if(row < numRows){
        unsigned int rowStart = rowPtrs[row];
        unsigned int rowEnd = rowPtrs[row+1];

        double rowSum = 0;
        for(int i = rowStart; i < rowEnd; i++){
            rowSum += (values[i] * ranks[colIdx[i]]);
        }
        newRanks[row] = 0.15 + 0.85 * rowSum;      //Damping factor added into the equation
    }
}

struct csrFormat{
    vector<int> rowPtrs;
    vector<int> colIdx;
    vector<double> values;

    int numNodes;
};

csrFormat *readFile(string fileName){
    csrFormat *csr = new csrFormat;

    ifstream inputFile;
    inputFile.open(fileName);

    string line;
    bool gotFirstLine = false;
    int numNodes;

    //also create a dictionary to map row to column
    cout << "Reading matrix from " << fileName << "..." << endl;
    map<int, vector<int>> node1_to_node2;
    map<int, vector<int>> node2_to_node1;
    while(getline(inputFile, line)){
        if(line.size() == 0 or line[0] == '%') continue;    //don't read comment lines

        if(!gotFirstLine){  //read the rows, cols, numvals from the first line in the file
            stringstream ss(line);
            string temp;
            ss >> temp;
            gotFirstLine = true;

            numNodes = stoi(temp);
            csr->numNodes = numNodes;
        } else {
            int node1;
            int node2;

            stringstream ss(line);
            ss >> node1;
            ss >> node2;

            node1_to_node2[node1-1].push_back(node2-1);
            node2_to_node1[node2-1].push_back(node1-1);
        }
    }
    inputFile.close();

    //create the correct rows vector in csr from rowCounts
    cout << "Populating CSR data structure..." << endl;

    csr->rowPtrs.push_back(0);  //initial index
    for(int i = 0; i < numNodes; i++){
        for(int j = 0; j < node1_to_node2[i].size(); j++){ //loop through the vector of columns with values in the current row
            csr->colIdx.push_back(node1_to_node2[i][j]);
            csr->values.push_back(1.0 / node2_to_node1[node1_to_node2[i][j]].size());  // 1 / number of out-edges
            // cout << csr->colIdx[csr->colIdx.size()-1] << " : " << csr->values[csr->values.size()-1] << endl;
        }
        csr->rowPtrs.push_back(csr->rowPtrs[i] + node1_to_node2[i].size());    //rowPtrs[i] = rowPtrs[i-1] + num values in row i
    }

    // for (auto list : node2_to_node1) {
    //     cout << list.first << ": ";
    //     for(auto v : list.second){
    //         cout << v << " ";
    //     }
    //     cout << endl;
    // }
    // cout << csr->rowPtrs.size() << endl;
    // cout << csr->colIdx.size() << endl;
    // cout << csr->values.size() << endl;

    return csr;
}

int main(int argc, const char ** argv){

    //User must provide a filename
    if(argc != 3){
        cout << "Error: requires a filename and a number of iterations" << endl;
        return -1;
    }

    //TIME TO READ IN GRAPH FROM FILE
    timespec start;
    timespec end;
    clock_gettime(CLOCK_MONOTONIC, &start);

    //Read in file from user input and get CSR struct
    csrFormat *csr = readFile(string(argv[1]));

    //END TIME TO READ IN FILE
    clock_gettime(CLOCK_MONOTONIC, &end);

    double diff = (1000000000 * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec) / (double)1000000000;
    cout << endl << "Time to read in file: " << diff << " seconds" << endl;

    /************ Pagerank code ************/

    ////////// Transfer data into CUDA kernel /////////////
    int* rowPtrs;
    int* colIdx;
    double* values;
    double* ranks;
    double* newRanks;
    //initialize ranks
    vector<double> ranksV(csr->numNodes, 10);
    vector<double> newRanksV(csr->numNodes, 10);

    int numIterations = atoi(argv[2]);

    hipMalloc((void **)&rowPtrs, sizeof(int) * csr->rowPtrs.size());
    hipMalloc((void **)&colIdx, sizeof(int) * csr->colIdx.size());
    hipMalloc((void **)&values, sizeof(double) * csr->values.size());
    hipMalloc((void **)&ranks, sizeof(double) * csr->numNodes);
    hipMalloc((void **)&newRanks, sizeof(double) * csr->numNodes);

    hipMemcpy(rowPtrs, csr->rowPtrs.data(), sizeof(int) * csr->rowPtrs.size(), hipMemcpyHostToDevice);
    hipMemcpy(colIdx, csr->colIdx.data(), sizeof(int) * csr->colIdx.size(), hipMemcpyHostToDevice);
    hipMemcpy(values, csr->values.data(), sizeof(double) * csr->values.size(), hipMemcpyHostToDevice);
    hipMemcpy(ranks, ranksV.data(), sizeof(double) * csr->numNodes, hipMemcpyHostToDevice);
    // cudaMemcpy(newRanks, newRanksV.data(), sizeof(double) * csr->numNodes, cudaMemcpyHostToDevice);

    //Time for pagerank algorithm
    clock_gettime(CLOCK_MONOTONIC, &start);

    //Call function
	int threadsPerBlock = 256;
	int blocksPerGrid = (1024 + threadsPerBlock - 1) / threadsPerBlock;

    int iter = 0;
    while(iter < numIterations){
        hipMemcpy(ranks, newRanksV.data(), sizeof(double) * csr->numNodes, hipMemcpyHostToDevice);
        pagerank<<<blocksPerGrid, threadsPerBlock>>>(rowPtrs, colIdx, values, ranks, newRanks, csr->numNodes);
        hipMemcpy(newRanksV.data(), newRanks, sizeof(double) * csr->numNodes, hipMemcpyDeviceToHost);  //copy the shared memory array back to the original vector

        iter++;
    }

    clock_gettime(CLOCK_MONOTONIC, &end);

    diff = (1000000000 * (end.tv_sec - start.tv_sec) + end.tv_nsec - start.tv_nsec) / (double)1000000000;
    cout << endl << "Time to Pagerank: " << diff << " seconds" << endl;

    hipFree(rowPtrs);
    hipFree(colIdx);
    hipFree(values);
    hipFree(ranks);
    hipFree(newRanks);


    ofstream textOut;
    textOut.open ("gpu_log.txt");
    for(int i = 0; i < csr->numNodes; i++){
        textOut << newRanksV[i] << " ";
        if(i % 20 == 0 && i != 0){
            textOut << endl;
        }
    }

    textOut.close();
}
















//
